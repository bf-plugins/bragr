#include "hip/hip_runtime.h"
#include <bifrost/xcorr_lite.h>
#include <bifrost/array.h>
#include <bifrost/common.h>
#include <bifrost/ring.h>
#include "cuda.hpp"
#include <utils.hpp>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>


#define DCP_DEBUG

extern "C" {

    __forceinline__ __device__
    void dp4a(int &c, const int &a, const int &b) {
        #if __CUDA_ARCH__ >= 610
          asm("dp4a.s32.s32 %0, %1, %2, %3;" : "+r"(c) : "r"(a), "r"(b), "r"(c)); 
        #else
          char4 &a4 = *((char4*)&a);
          char4 &b4 = *((char4*)&b);
          c += a4.x*b4.x;
          c += a4.y*b4.y;
          c += a4.z*b4.z;
          c += a4.w*b4.w;
        #endif
        }

    /*
      cmult_dp4a -- Do complex conjugate multiply accumulate <A*Conj(B)>
      Using two dp4a instructions. Takes 8-bit complex data 
      packed as a single 32-bit int [8R8I 8R8I]. 
  
      For two complex numbers:
          ab* = (ar + i*ai)(br + i*bi)
          re(ab*) = ar*br + ai*bi
          im(ab*) = ai*br - ar*bi
      So use two dp4a to compute:
          [a0r a0i a1r a1i].[b0r b0i b1r b1i]   = Re(<ab*>)
          [a0r a0i a1r a1i].[-b0i b0r -b1i b1r] = Im(<ab*>)
      Where angled brackets denote time averaging (over 2x samples)
    */
    __forceinline__ __device__
    void cmult_dp4a(int &res_re, int &res_im, int &A, int &B) {
        // Unpack 32-bit int into 8-bit
        int8_t Bmod[4];
        int8_t *b8 = (int8_t *)&B;      
    
        // Transpose for bmod 
        Bmod[0] = -b8[1];
        Bmod[1] = b8[0];
        Bmod[2] = -b8[3];
        Bmod[3] = b8[2]; 
    
        //int8_t *a8 = (int8_t *)&A;
        //printf("A %d %d %d %d | B %d %d %d %d\\n", a8[0], a8[1], a8[2], a8[3], b8[0], b8[1], b8[2], b8[3]);
    
        // Pack 8-bit to 32-bit
        int &Bmodp = *((int *)&Bmod); 
    
        // Run complex multiply
        dp4a(res_re, A, B);
        dp4a(res_im, A, Bmodp);
        }


    // Data array should be (heap H, frequency F, antenna N, fine_time T)
    __global__ void xcorrDp4aKernel
        (int *data, float *xcorr, int H, int N, int T, int reset)
        {
        // Setup thread indexes
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        // note that F(nchan) is gridDim.z
        
        // Only run if antenna IDs X and Y are < N_antenna (N)
        if (x < N && y < N) {

            // Loop over heap (H)
            for (int h = 0; h < H; h++) {

                int chan_offset_out = blockIdx.z * N * N * 2;
                int ant_offset      = T / 2;  //x2 for complex, but /4 for packed
                int heap_offset_out = h * gridDim.z * N * N * 2;

                //                        heap_offset              +  chan_offset
                int heapchan_offset_in = (h * gridDim.z * N * T/2) + (blockIdx.z * N * T/2);
                
                int xy_real = 0;
                int xy_imag = 0;
                const int idx = 2*y + N*2*x + chan_offset_out + heap_offset_out; // Compute index for output array
                
                // Note -- using dp4a must be careful of bit growth.
                // output of each 8-bit dot product is 16 bits
                // Adding 4x 16-bit numbers = 18-bit number
                // accumulator is only 32 bits, so using 18 of 32 bits.
                // Max 14 bits of growth = 2^14 = 4096 integrations
                
                const int ia  = heapchan_offset_in + ant_offset*x;
                const int ib  = heapchan_offset_in + ant_offset*y;
               
                for (int t = 0; t < T/2; t++) {         
                    //printf("idx %d | x%d.y%d | A %dx%d\\n", idx, x, y, ia + t, ib + t);
                    //cmult_dp4a(xcorr[idx], xcorr[idx+1], data[ia + t], data[ib + t]);
                    cmult_dp4a(xy_real, xy_imag, data[ia + t], data[ib + t]);
                }
            
                // Copy xy* result to device mem
                if (reset != 0) {
                    xcorr[idx]   = (float) xy_real;
                    xcorr[idx+1] = (float) xy_imag;
                } else {
                    xcorr[idx]   += (float) xy_real;
                    xcorr[idx+1] += (float) xy_imag;
                }
              } // Loop over heap (H)
           } // if x < N and y < N
        }
        
    void launch_xcorr_lite(int *data, float *xcorr, int H, int F, int N, int T, int reset) {
        dim3 block, grid;
        int GS = 1; // Grid size for antenna
        int NB = N;     // Block size for antenna
        // Set grid size for N > 32
        if(N > 32) {
            GS = ceil(float(N) / 32);
            NB = 32;
        }

        grid.x = GS;
        grid.y = GS;
        grid.z = F;
        
        block.x = NB;
        block.y = NB;
        block.z = 1;

        hipStream_t g_cuda_stream = 0;

#ifdef DCP_DEBUG
        //printf("N: %d SHM %d\n", N, shm_bytes);
  printf("Debug: <<<B: (%d, %d, %d) G: (%d, %d, %d)  >>>\n", block.x,
         block.y, block.z, grid.x, grid.y, grid.z);
#endif
         int shm = 0;
        xcorrDp4aKernel<<< grid, block, shm, g_cuda_stream >>>(data, xcorr, H, N, T, reset);
    }
        


    BFstatus XcorrLite(BFarray *bf_data, BFarray *bf_xcorr, int reset)
    {
        
        int* data = (int *)bf_data->data;
        float* xcorr = (float *)bf_xcorr->data;

        int H = bf_data->shape[0]; // Heap (slow time axis)
        int F = bf_data->shape[1]; // Frequency
        int N = bf_data->shape[2]; // Antenna
        int T = bf_data->shape[3]; // Fine time
        
        //printf("ispan dims F: %d N: %d T: %d\n", F, N, T);
        launch_xcorr_lite(data, xcorr, H, F, N, T, reset);
        
        BF_CHECK_CUDA(hipGetLastError(), BF_STATUS_DEVICE_ERROR);

        return BF_STATUS_SUCCESS;
    }

}