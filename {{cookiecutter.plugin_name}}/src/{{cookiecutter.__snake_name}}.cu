#include <bifrost/xcorr_lite.h>
#include <bifrost/array.h>
#include <bifrost/common.h>
#include <bifrost/ring.h>
#include "cuda.hpp"
#include <utils.hpp>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include "{{cookiecutter.__snake_name}}.h"
thread_local hipStream_t g_cuda_stream = hipStreamPerThread;

/*!
  \class {{cookiecutter.__class_name}}
  \brief Plugger class for {{cookiecutter.__camel_name}}
*/
class {{cookiecutter.__class_name}} {
private:
    // Parameters passed during init() may be stored here, use _n_param 
    int _n_param1;
    hipStream_t _stream;
    
public:
    {{cookiecutter.__class_name}}() : _stream(g_cuda_stream) {}
    ~{{cookiecutter.__class_name}}() {
        hipDeviceSynchronize();
    }
    
    inline int n_param1() const { return _n_param1; }

    // Initialize your plugin here
    void init(int n_param1) {
        _n_param1 = n_param1; // This stores parameter as a private _n_params
       
        // Zero out any existing state
        this->reset_state();
    }

    // Needed to set CUDA stream for asynchronous launching
    void set_stream(hipStream_t stream) {
        hipDeviceSynchronize();
        _stream = stream;
    }

    // Do any zeroing / memset stuff here
    void reset_state() {
        
    }

    // execute your plugin
    void exec(BFarray const* in, BFarray* out) {
        
        // Check for errors
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);  
        }
};

// Used by bifrost python wrapper at instantiation
BFstatus {{cookiecutter.__camel_name}}Create(bfplugin* plan_ptr) {
    BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN_ELSE(*plan_ptr = new {{cookiecutter.__class_name}}(),
                       *plan_ptr = 0);
}

// Initialisation for plugin 
BFstatus {{cookiecutter.__camel_name}}Init(bfplugin plan, int n_param1) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_TRY_RETURN(plan->init(n_param1));
}

// Assign to CUDA stream
BFstatus {{cookiecutter.__camel_name}}SetStream(bfplugin plan, void const* stream) {
        BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
        BF_ASSERT(stream, BF_STATUS_INVALID_POINTER);
        BF_TRY_RETURN(plan->set_stream(*(hipStream_t*)stream));
}

// Reset state of any internal memory 
BFstatus {{cookiecutter.__camel_name}}ResetState(bfplugin plan) {
        BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
        BF_TRY_RETURN(plan->reset_state());
}

// Main method to execute data processing tasks
BFstatus {{cookiecutter.__camel_name}}Execute(bfplugin plan,
                     BFarray const* in,
                     BFarray*       out) {
    BF_ASSERT(plan, BF_STATUS_INVALID_POINTER);
    BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
  	BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
    
    BF_ASSERT(space_accessible_from(in->space, BF_SPACE_CUDA),
              BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA),
              BF_STATUS_UNSUPPORTED_SPACE);
    
    BF_TRY_RETURN(plan->exec(in, out));
}

// Called by python wrapper at deletion time
BFstatus {{cookiecutter.__camel_name}}Destroy(bfplugin plan) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    delete plan;
    return BF_STATUS_SUCCESS;
}